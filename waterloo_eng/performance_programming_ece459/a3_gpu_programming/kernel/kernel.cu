
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void conv_layer(double input[100][100], double filter[10][5][5], double conv_output[10][20][20]) {

	int blockId = blockIdx.x; // size 10
	int threadId = threadIdx.x; // size 400

	// Preform the element-wise matrix multiplication
	double product = 0.0;
	for (int i = 0; i < 5; i++) {
        	for (int j = 0; j < 5; j++) {
            		product += input[(5*(threadId/20)) + i][(5*(threadId%20)) + j] * filter[blockId][i][j];
        	}
    	}

	// perform relu
	if (product < 0){
		product = 0;
	}

	// Store results in conv output layer
	conv_output[blockId][threadId/20][(threadId%20)] = product;
}

extern "C" __global__ void out_layer(double input[4000], double weights[10][4000], double output[10]) {

	int threadId = threadIdx.x; // size 10

	// Multiply conv output by each set of weights
	// Each thread calculates the dot product for that neuron
	for (int i = 0; i < 4000; i++){
		output[threadId] += input[i] * weights[threadId][i];
	}
}
